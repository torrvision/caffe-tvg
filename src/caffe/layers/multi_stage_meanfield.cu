#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/layers/meanfield_layers.hpp"

#include "caffe/util/tvg_ho_utils.hpp"

namespace caffe {

// Avoid divergence by uncoalescing access
    template <typename Dtype>
    __global__ void  computeBilateralKernel(const  int num_pixels_,
                                            const Dtype* const rgb_blob,
                                            const int width_, const int height_, const int channels_,
                                            float theta_alpha_, float theta_beta_,
                                            const int n, float* const output_kernel) {
        int offset = ((n * channels_ ) * height_) * width_ ;
        CUDA_KERNEL_LOOP(p, num_pixels_) {
            output_kernel[5 * p] = (float)(p % width_) / theta_alpha_;
            output_kernel[5 * p + 1] = (float)(p / width_) / theta_alpha_;
            const Dtype * const rgb_data_start = rgb_blob + offset;
            output_kernel[5 * p + 2] = (float)(rgb_data_start[p] / theta_beta_);
            output_kernel[5 * p + 3] = (float)((rgb_data_start + num_pixels_)[p] / theta_beta_);
            output_kernel[5 * p + 4] = (float)((rgb_data_start + num_pixels_ * 2)[p] / theta_beta_);
        }
    }

    template <typename Dtype>
    __global__ void computeNorm(Dtype* norm_output_data, int num_pixels){
        CUDA_KERNEL_LOOP(i, num_pixels) {
            norm_output_data[i] = 1.f / (norm_output_data[i] + 1e-20f);
        }
    }

/**
 * Performs filter-based mean field inference given the image and unaries.
 *
 * bottom[0] - Unary terms
 * bottom[1] - Softmax input/Output from the previous iteration (a copy of the unary terms if this is the first stage).
 * bottom[2] - RGB images
 * bottom[3] - Indices used for reading detection and superpixel files
 *
 * top[0] - Output of the mean field inference (not normalized).
 */
    template <typename Dtype>
    void MultiStageMeanfieldLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

        if (init_cpu_ && this->layer_param().multi_stage_meanfield_param().force_cpu()){
            this->Forward_cpu(bottom, top);
            return;
        }

        if(init_cpu_) {
            LOG(FATAL)
            << ("You initialised your network on CPU, please initialise it on GPU.");
        }

        if (detection_potentials_enabled_) {
          init_detections(static_cast<int>(bottom[3]->cpu_data()[0]));
        }

        const Dtype* bottom_data = bottom[2]->gpu_data() ; // The RGB image
        split_layer_bottom_vec_[0] = bottom[0];
        split_layer_->Forward(split_layer_bottom_vec_, split_layer_top_vec_);

        // Initialize the bilateral lattices.
        computeBilateralKernel<Dtype><<<CAFFE_GET_BLOCKS(num_pixels_), CAFFE_CUDA_NUM_THREADS>>>(
            num_pixels_, bottom_data, width_, height_, channels_,
            theta_alpha_, theta_beta_, 0,
            bilateral_kernel_buffer_); // The 0 refers to the offset (which is the case for a batch size of 1

        CUDA_POST_KERNEL_CHECK;
        bilateral_lattice_.reset(new ModifiedPermutohedral());
        bilateral_lattice_->init_gpu(bilateral_kernel_buffer_, 5, width_, height_);
        // Calculate bilateral filter normalization factors.
        Dtype* norm_output_data = bilateral_norms_.mutable_gpu_data();
        bilateral_lattice_->compute_gpu(norm_output_data, norm_feed_, 1);
        computeNorm<Dtype><<<CAFFE_GET_BLOCKS(num_pixels_), CAFFE_CUDA_NUM_THREADS>>>(norm_output_data, num_pixels_);
        CUDA_POST_KERNEL_CHECK;

        for (int i = 0; i < num_iterations_; ++i) {
            meanfield_iterations_[i]->PrePass(this->blobs_, bilateral_lattice_, &bilateral_norms_);

            if (is_no_class_weights_){
            meanfield_iterations_[i]->InitLocalCompatibility(num_, channels_, height_, width_);
            meanfield_iterations_[i]->Reshape(num_, channels_, height_, width_);
            }

            meanfield_iterations_[i]->Forward_gpu();
        }


        // Output the latent Y variables
        // Actually easier to do this on CPU, since that is where the data is updated
        Dtype * const y_top_data = top[1]->mutable_cpu_data();
        caffe_set(top[1]->count(), Dtype(0.), y_top_data);

        if (detection_potentials_enabled_) {
          const Dtype *const inferred_y_data = detection_y_qs_[num_iterations_]->cpu_data();
          const int aval_y_data_count = detection_y_qs_[num_iterations_]->count();
          for (int i = 0; i < aval_y_data_count; ++i) {
            y_top_data[i] = inferred_y_data[i];
          }
        }
    }

/**
 * Backprop through filter-based mean field inference.
 */

template<typename Dtype>
void MultiStageMeanfieldLayer<Dtype>::Backward_gpu(
        const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
        const vector<Blob<Dtype>*>& bottom) {

    if (init_cpu_ && this->layer_param().multi_stage_meanfield_param().force_cpu()){
        this->Backward_cpu(bottom, propagate_down, top);
        return;
    }

    if(init_cpu_){
        LOG(FATAL) << ("You initialize your network on CPU, please initialize it on GPU.");
    }

    if (detection_potentials_enabled_ && detection_count_ > 0) {
      caffe_set(detection_y_qs_[num_iterations_]->count(), Dtype(0.), detection_y_qs_[num_iterations_]->mutable_cpu_diff());
    }

    for (int i = (num_iterations_ - 1); i >= 0; --i) {
        meanfield_iterations_[i]->Backward_gpu();
    }

    vector<bool> split_layer_propagate_down(1, true);
    split_layer_->Backward(split_layer_top_vec_, split_layer_propagate_down, split_layer_bottom_vec_);

    // Accumulate diffs from mean field iterations.
    for (int blob_id = 0; blob_id < this->blobs_.size(); ++blob_id) {

        Blob<Dtype>* cur_blob = this->blobs_[blob_id].get();

        if (this->param_propagate_down_[blob_id]) {

            caffe_gpu_set(cur_blob->count(), Dtype(0), cur_blob->mutable_gpu_diff());

            for (int i = 0; i < num_iterations_; ++i) {
                const Dtype* diffs_to_add = meanfield_iterations_[i]->blobs()[blob_id]->gpu_diff();
                caffe_gpu_axpy(cur_blob->count(), Dtype(1.), diffs_to_add, cur_blob->mutable_gpu_diff());
            }
        }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiStageMeanfieldLayer);

}  // namespace caffe